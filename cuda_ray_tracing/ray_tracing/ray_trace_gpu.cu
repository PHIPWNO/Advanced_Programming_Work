#include "hip/hip_runtime.h"
extern "C" {
#include "bitmap.h"
}
#include"hip/hip_runtime_api.h"
#include<math.h>
#include<stdlib.h>
#include<stdio.h>


#define OUTFILE "output.bmp"

__global__ void bw_kernel(RgbTriple* img, int n, int x_stride, int y_stride)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float L[3] = {4,4,-1}, C[3]={0,12,0}, I[3] = {0,0,0};
	float N[3], S[3];
	float R = 4.4, W_max = 10, delta_x = 2 * W_max / (float) n;
	float delta_z = delta_x;
	int b;


	for (int i = x; i < n; i += x_stride) {
		for (int j = y; j < n; j += y_stride) {
			float W_x = i * delta_x - W_max;
			float W_z = j * delta_z - W_max;
			float D = R * R - (W_x - C[0]) * (W_x - C[0]) - 
			(W_z - C[2]) * (W_z - C[2]);
			if(D < 0){
				b = 0;
			}
			else{
				I[1] = C[1] - sqrtf(D);
				I[0] = W_x; I[2] = W_z;
				float I_min_C_norm = sqrtf(powf(I[0]-C[0],2) + powf(I[1]-C[1],2) + powf(I[2]-C[2],2));
				float L_min_I_norm = sqrtf(powf(L[0]-I[0],2) + powf(L[1]-I[1],2) + powf(L[2]-I[2],2));
				N[0] = (I[0] - C[0]) / I_min_C_norm; N[1] = (I[1] - C[1]) / I_min_C_norm;
				N[2] = (I[2] - C[2]) / I_min_C_norm;
				S[0] = (L[0] - I[0]) / L_min_I_norm; S[1] = (L[1] - I[1]) / L_min_I_norm;
				S[2] = (L[2] - I[2]) / L_min_I_norm;
				float dot_prod = N[0] * S[0] + N[1] * S[1] + N[2] * S[2];
				if (dot_prod > 0){
					b = (int) floorf(dot_prod * 255); 
					//b = 170;
				}
				else{
					b = 0;
				}
			}

			RgbTriple* p = &img[i * n + j];
			p->red = b;
			p->green = b;
			p->blue = b;
		}
	}

	return;
}

int main(int argc, char* argv[])
{
	int image_size, grid_dim_x, grid_dim_y, block_dim_x, block_dim_y, n;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
		
	if (argc < 6) {
		fprintf(stderr, "5 arguments in this order:  image size,  grid dim x,"
		" grid dim y, block dim x, and block dim y\n");
		exit(EXIT_FAILURE);
	}

	n = atoi(argv[1]);
	image_size = n * n;
	grid_dim_x = atoi(argv[2]); grid_dim_y = atoi(argv[3]);
	block_dim_x = atoi(argv[4]); block_dim_y = atoi(argv[5]);

	if(prop.maxThreadsPerBlock < block_dim_x * block_dim_y){
		fprintf(stderr, "illegal block dimension!\n");
		return -1;
	}

	hipEvent_t tick, tock;
	int x_stride = grid_dim_x * block_dim_x;
	int y_stride = grid_dim_y * block_dim_y;


	/* allocate memory for bmp on host */
	RgbTriple* img = (RgbTriple*) malloc(image_size * sizeof(RgbTriple));

	/* allocates GPU memory for bmp*/
	RgbTriple* d_img;
	checkCudaErrors(hipMalloc((void**)&d_img, sizeof(RgbTriple) * image_size));
	checkCudaErrors(hipMemcpy(d_img, img, sizeof(RgbTriple) * image_size, hipMemcpyHostToDevice));

	/* Set gridDim and blockDim */
	dim3 blockDim(block_dim_x, block_dim_y, 1);
	dim3 gridDim(grid_dim_x, grid_dim_y, 1);


	hipEventCreate(&tick);
	hipEventCreate(&tock);

	hipEventRecord(tick, 0);

	/* how to check kernel errors
	https://developer.nvidia.com/blog/how-query-device-properties-and-handle-errors-cuda-cc/
	*/
	bw_kernel<<<gridDim, blockDim>>>(d_img, n, x_stride, y_stride);

	/* stop timing */
	hipEventRecord(tock, 0);
	hipEventSynchronize(tock);

	hipError_t errSync  = hipGetLastError(); //gets last error off stack
hipError_t errAsync = hipDeviceSynchronize(); //This is because of asynchornicity off kernel 

if (errSync != hipSuccess){
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  return -1;
}
if (errAsync != hipSuccess){
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}


	float time;
	hipEventElapsedTime(&time, tick, tock);
	hipEventDestroy(tick); hipEventDestroy(tock);
	fprintf(stderr, "runtime: %f seconds \t%f ms\n", time / 1000000, time);
	printf("%f\n", time);

	// Get img from device and write it to file
	checkCudaErrors(hipMemcpy(img, d_img, sizeof(RgbTriple) * image_size, hipMemcpyDeviceToHost));
	
	save_bitmap(img, n, n, OUTFILE);

	free(img);
	checkCudaErrors(hipFree(d_img));
	
	
}
